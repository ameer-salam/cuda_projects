#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <random>

__global__ void array_summation(int *a, int *b, int *c, int size)
{
	int gid = threadIdx.x + (blockDim.x * blockIdx.x);
	if (gid < size)
		c[gid] = a[gid] + b[gid];
}

int main()
{
	int array_size = 10; 
	int array_byte_size = array_size * sizeof(int);
	int *a1, *a2, *result;

	//allocating memory space for the array1, array2 and reult
	a1 = (int*)malloc(array_byte_size);
	a2 = (int*)malloc(array_byte_size);
	result = (int*)malloc(array_byte_size);

	//assigning values to the array space
	for (int i = 0; i < array_size; i++)
	{
		a1[i] = rand() % 100;
		a2[i] = rand() % 100;
	}
	printf("The randomly choosen elements are : \n");
	for (int i = 0; i < array_size; i++)
		printf("a1[%d] : %d\ta2[%d] : %d\n", i, a1[i], i, a2[i]);

	//declaring device variables and allocing memory for the same
	int *d_a1, *d_a2, *d_result;
	hipMalloc((int**)&d_a1, array_byte_size);
	hipMalloc((int**)&d_a2, array_byte_size);
	hipMalloc((int**)&d_result, array_byte_size);

	//copying the values form the host to device variables
	hipMemcpy(d_a1, a1, array_byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_a2, a2, array_byte_size, hipMemcpyHostToDevice);

	array_summation << 1, array_size >> (&d_a1, &d_a2, &d_result, array_size);

	hipDeviceSynchronize();

	hipMemcpy(d_result, result, array_byte_size, hipMemcpyHostToDevice);
	
	for (int i = 0; i < array_size; i++)
		printf("a1[%d] + a1[%d] = %d\n", i, i, result[i]);
	
	hipDeviceReset();
	return 0;
}