#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void array_summation(int *a, int *b, int *c, int size)
{
	int gid = threadIdx.x + (blockDim.x * blockIdx.x);
	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

//int main()
//{
//	int array_size = 10;
//	int array_byte_size = array_size * sizeof(int);
//	
//	//declaring the host variables
//	int *a1, *a2, *result;
//	
//	//allocating memory space for a1, a2 and results variables
//	a1 = (int*)malloc(array_byte_size);
//	a2 = (int*)malloc(array_byte_size);
//	result = (int*)malloc(array_byte_size);
//
//	//generating random elements to feed into the a1 and a2 arrays
//	for (int i = 0; i < array_size; i++)
//	{
//		a1[i] = rand() % 100;
//		a2[i] = rand() % 100;
//	}
//	
//	//printing the arrays
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("a1[%d] : %d\ta2[%d] : %d\n", i, a1[i], i, a2[i]);
//	}
//
//	//declaring pointer variabled to store the address of the device variables to access them
//	int *device_a1_pointer, *device_a2_pointer, *device_result_pointer;
//	
//	//allocating memory to hold the arrays and result in the device and returning the memory sequence pointers to the pointer variables above
//	cudaMalloc((int**)&device_a1_pointer, array_byte_size);
//	cudaMalloc((int**)&device_a2_pointer, array_byte_size);
//	cudaMalloc((int**)&device_result_pointer, array_byte_size);
//
//	//copying the data from the host variables to the device variables
//	cudaMemcpy(device_a1_pointer, a1, array_byte_size, cudaMemcpyHostToDevice);
//	cudaMemcpy(device_a2_pointer, a2, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 grid(1);
//	dim3 block(array_size);
//
//	//calling the kernel
//	array_summation << <grid, block >> > (device_a1_pointer, device_a2_pointer, device_result_pointer, array_size);
//	cudaDeviceSynchronize();
//	
//	//after the kernel has executed copy the result from device to the host
//	cudaMemcpy(result, device_result_pointer, array_byte_size, cudaMemcpyDeviceToHost);
//
//	for (int i = 0; i < array_size; i++)
//		printf("a1[%d] + a2[%d] = %d\n", i, i, result[i]);
//
//	free(a1);
//	free(a2);
//	free(result);
//	free(device_a1_pointer);
//	free(device_a2_pointer);
//	free(device_result_pointer);
//	cudaDeviceReset();
//	return 0;
//}