#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void uniqueIdx_call_threadIdx(int* input) //the aprameter is the pointer to the integer array
{
	int tid = threadIdx.x;
	printf("threadIdx: \t%d value: \t%d\n", tid, input[tid]);
}

//int main()
//{
//	int array_size = 8;
//	int array_byte_size = sizeof(int) * array_size;
//	int data[] = { 45,3,9,12,4,0,81,32 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("%d  ", data[i]);
//	}
//	printf("\n");
//
//	int * a_data;
//	cudaMalloc((void**)&a_data, array_byte_size);
//	cudaMemcpy(a_data, data, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 grid(1);
//	dim3 block(8);
//	uniqueIdx_call_threadIdx << <grid, block >> > (a_data);
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}