#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void rando_number_(int* input)
{
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);
	printf("threadIdx.x : %d.\tblockIdx.x : %d,\tgid : %d\tValue : %d\n", threadIdx.x, blockIdx.x, gid, input[gid]);
}

//int main()
//{
//	int size = 128;
//	int byte_size = size * sizeof(int);
//
//	int* h_data;
//	h_data = (int*)malloc(byte_size);
//
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < size; i++)
//		h_data[i] = (int)(rand() & 0xff);
//	int* d_input;
//	cudaMalloc((void**)&d_input, byte_size);
//	cudaMemcpy(d_input, h_data, byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(64);
//	dim3 grid(2);
//
//	rando_number_ << <grid, block>> > (d_input);
//
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//
//	return 0;
//}
//
