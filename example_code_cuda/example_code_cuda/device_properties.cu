
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include<stdio.h>
//#include<stdlib.h>
//#include<random>
//
//
//
//int main()
//{
//	int device_count = 0; //variable to store the number of cuda enabled devices on the system
//	cudaGetDeviceCount(&device_count);
//	if (device_count == 0)
//		printf("No CUDA enabled devices found!\n");
//	else
//		printf("The number of CUDA enables devices avaiable are : %d\n", device_count);
//
//
//	//cuda get device properties function
//	int dev_no = 0;
//	cudaDeviceProp dProp;
//	cudaGetDeviceProperties(&dProp, dev_no);
//
//	//printing the device properties
//	printf("Device Name : %s\n", dProp.name);
//	printf("Number of multiprocessors : %d\n", dProp.multiProcessorCount);
//	printf("Clock rate : %d\n", dProp.clockRate);
//	printf("Compute capabillities : %d.%d kb\n", dProp.major, dProp.minor);
//	printf("Total Global Memory : %4.4f kb\n", dProp.totalGlobalMem/1024.0);
//	printf("Total Constant Memory : %4.4f kb\n", dProp.totalConstMem/1024.0);
//	printf("Total Shared memory per block : %4.4f kb\n", dProp.sharedMemPerBlock/1024.0);
//	printf("Total Shated memory per MP : %4.4f kb\n", dProp.sharedMemPerMultiprocessor / 1024.0);
//	printf("Total number of registers available per block : %d\n", dProp.regsPerBlock);
//	printf("Total number of registers available per MP : %d\n", dProp.regsPerMultiprocessor);
//	printf("Total number of threads available per block : %d\n", dProp.maxThreadsPerBlock);
//	printf("Total number of registers available per MP : %d\n", dProp.maxThreadsPerMultiProcessor);
//	printf("Wrap size : %d\n", dProp.warpSize);
//	printf("Maximum grid size : %d\n", dProp.maxGridSize);
//	printf("Maximum block dimension : %d\n", dProp.maxThreadsDim);
//	cudaDeviceReset();
//	return 0;
//}