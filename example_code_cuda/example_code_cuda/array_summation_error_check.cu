#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <random>

__global__ void array_summation_2(int *a, int *b, int *c, int size)
{
	int gid = threadIdx.x + (blockDim.x * blockIdx.x);
	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

void array_summation_2_cpu(int *a, int *b, int *c, int size)
{
	printf("CPU Summation function called!\n");
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
	//below statement is just for validation
	//printf("CPU Summation of a1[998] : %d\t + a2[998] : %d\t is = %d\n", a[998], b[998], c[998]);
}

void array_summation_2_validation(int *a, int *b, int size)
{
	printf("validation function called!\n");
	for (int i = 0; i < size; i++)
	{
		if (a[i] != b[i])
		{
			printf("They do not match!\b");
			return;
		}
	}
	printf("\n\nArrays match! and have been validated!\n");
}

//int main()
//{
//	int array_size = 1000;
//	int array_byte_size = array_size * sizeof(int);
//	int *a1, *a2, *result, *c_results;
//	
//	//declaring variable of type cudaError to store the error value
//	cudaError error;
//	
//	//allocating memory space for the above variables a1, a2, result
//	a1 = (int*)malloc(array_byte_size);
//	a2 = (int*)malloc(array_byte_size);
//	result = (int*)malloc(array_byte_size);
//
//	//generating random numbers and storing them in the above array
//	for (int i = 0; i < array_size; i++)
//	{
//		a1[i] = rand() % 100 + 1;
//		a2[i] = rand() % 100 + 1;
//	}
//
//	//printing the randomly generated array elements
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("a1[%d] : %d\ta2[%d] : %d\n", i, a1[i], i, a2[i]);
//	}
//
//	//declaring variables to store the memory addresses of the device variables
//	int *device_a1_address, *device_a2_address, *device_result_address;
//
//	//allocating the memory in the device and storing the memory in the above pointer variables
//	//we will also assign the return of the memory allocated function to the cudaerror variables
//	error = cudaMalloc((int**)&device_a1_address, array_byte_size);
//	if (error != cudaSuccess)
//		fprintf(stderr, "%s\n", cudaGetErrorString(error));
//
//	error = cudaMalloc((int**)&device_a2_address, array_byte_size);
//	if (error != cudaSuccess)
//		fprintf(stderr, "%s\n", cudaGetErrorString(error));
//
//	error = cudaMalloc((int**)&device_result_address, array_byte_size);
//	if (error != cudaSuccess)
//		fprintf(stderr, "%s\n", cudaGetErrorString(error));
//
//	//after decleration of the array, copying the values from the host variable to the device variables
//	cudaMemcpy(device_a1_address, a1, array_byte_size, cudaMemcpyHostToDevice);
//	cudaMemcpy(device_a2_address, a2, array_byte_size, cudaMemcpyHostToDevice);
//
//	//declaring the grid and block variables
//	int block_size = 128;
//	dim3 block(block_size);
//	dim3 grid((array_size/block_size+1));
//
//	//launch of the kernel function
//	array_summation_2 <<< grid, block>>> (device_a1_address, device_a2_address, device_result_address, array_byte_size);
//	cudaDeviceSynchronize();
//	cudaMemcpy(result, device_result_address, array_byte_size, cudaMemcpyDeviceToHost);
//
//	//validate the cuda generated result by CPU generated result
//	c_results = (int*)malloc(array_byte_size);
//	array_summation_2_cpu(a1, a2, c_results, array_size);
//
//	//validation function
//	array_summation_2_validation(c_results, result, array_size);
//
//	return 0;
//}