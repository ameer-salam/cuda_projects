#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <random>

//cpu addition function
void cpu_addition(int* a, int* b, int* c, int* results, int size)
{
	for (int i = 0; i < size; i++)
	{
		results[i] = a[i] + b[i] + c[i];	
	}
	printf("CPU calculations done!\n");
}

//gpu addition function
__global__ void gpu_addition(int *a, int *b, int *c, int *d, int size)
{
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (gid < size)
	{
		d[gid] = a[gid] + b[gid] + c[gid];
	}
}

void compare(int *a, int *b, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (a[i] != b[i])
		{
			printf("They do not match!");
			return;
		}
			
	}
	printf("\nThe arrays Match!\n\n");
	int randno = rand() % size + 1;
	printf("Example : cpu_result[%d] = %d\tgpu_result[%d] = %d\n", randno, a[randno], randno, b[randno]);
	randno = rand() % size + 1;
	printf("Example : cpu_result[%d] = %d\tgpu_result[%d] = %d\n", randno, a[randno], randno, b[randno]);
}

int main()
{
	int array_size = 4194304;
	int array_byte_size = 4194304 * sizeof(int);
	int *a1, *a2, *a3 ,*c_result, *g_results;

	//allocating memeory for the array1 and array2
	a1 = (int*)malloc(array_byte_size);
	a2 = (int*)malloc(array_byte_size);
	a3 = (int*)malloc(array_byte_size);
	c_result = (int*)malloc(array_byte_size);
	g_results = (int*)malloc(array_byte_size);

	//randomly allocating elements to the  arays
	for (int i = 0; i < array_size; i++)
	{
		a1[i] = rand() % 1000 + 1;
		a2[i] = rand() % 1000 + 1;
		a3[i] = rand() % 1000 + 1;
	}

	//cpu addition function
	clock_t cpu_clock_start, cpu_clock_stop;
	cpu_clock_start = clock();
	cpu_addition(a1, a2, a3, c_result, array_size);
	cpu_clock_stop = clock();

	//preparing for the kernel function
	//declaring pointer variables to point to the device memory
	int *d_a1, *d_a2, *d_a3, *d_results;

	//allocating memory to store the variables
	hipMalloc((int**)&d_a1, array_byte_size);
	hipMalloc((int**)&d_a2, array_byte_size);
	hipMalloc((int**)&d_a3, array_byte_size);
	hipMalloc((int**)&d_results, array_byte_size);

	//transfering data from host to the device
	hipMemcpy(d_a1, a1, array_byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_a2, a2, array_byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_a3, a3, array_byte_size, hipMemcpyHostToDevice);

	//declaring the grid and block
	int block_size = 128;
	dim3 block(block_size);
	dim3 grid(array_size / block_size);

	//calling the kernel function
	clock_t gpu_clock_start, gpu_clock_stop;
	gpu_clock_start = clock();
 	gpu_addition << <grid, block>> > (d_a1, d_a2, d_a3, d_results, array_size);
	hipDeviceSynchronize();
	gpu_clock_stop = clock();

	//copying back the result from GPU to CPU
	hipMemcpy(g_results, d_results, array_byte_size, hipMemcpyDeviceToHost);


	//comparission function
	compare(c_result, g_results, array_size);

	free(a1); free(a2); free(a3); free(c_result); free(g_results);
	hipDeviceReset();

	return 0;
}