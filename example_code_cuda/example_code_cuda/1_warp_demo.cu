
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <random>
//
//__global__ void warp_demo()
//{
//	int gid = threadIdx.x + (blockIdx.x * blockDim.x) +
//		blockIdx.y + (gridDim.x * blockDim.x);
//	int warp_id = gid / 32;
//	int g_b_id = blockIdx.y * gridDim.x + blockIdx.x;
//	printf("threadIdx.x : %d\tblockIdx.x : %d\tblockIdx.y : %d\tgid : %d\twarp_id : %d\tgbid : %d\n",
//		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, g_b_id);
//
//}
//
//int main()
//{
//	dim3 block(42);
//	dim3 grid(2,2);
//	warp_demo << <grid, block>> > ();
//	return 0;
//}