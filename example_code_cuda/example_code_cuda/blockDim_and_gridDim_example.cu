#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void blockAndGrid_dim_example()
{
	printf("blockDim.x : %d\tblockDim.y : %d\tblockDim.z : %d\tgridDim.x : %d\tgridDim.y : %d\tgridDim.z : %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

//int main()
//{
//	int nx, ny;
//	nx = 16, ny = 16;
//	dim3 block(8, 8);
//	dim3 grid(nx / block.x, ny / block.y);
//	blockAndGrid_dim_example << <grid, block >> > ();
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}