#include "hip/hip_runtime.h"


#include <stdio.h>

//this is for single dimensional grid
__global__ void thread_excersise_(int* input)
{
	int tid = threadIdx.x;
	int index = tid + (blockIdx.x * blockDim.x);
	printf("threadIdx.x : %d,\t, blockIdx.x : %d,\t, value : %d\n", tid, blockIdx.x, input[index]);
}

//for two dimensional grid
__global__ void thread_excersise_2d(int* input)
{
	int tid = threadIdx.x;
	//to calculate the number of threads in a row -> gridDim.x * blockDim.x
	int row_offset = (gridDim.x * blockDim.x) * blockIdx.y;
	int col_offset = (blockIdx.x * blockDim.x);
	int gid = tid + (col_offset + row_offset);
	printf("threadIdx.x : %d,\t, blockIdx.x : %d,\tblockIdx.y : %d,\tgid : %d,\tvalue : %d\n", tid, blockIdx.x, blockIdx.y, gid, input[gid]);
}

//int main()
//{
//	int array[] = { 77,43,5,3,2,8,34,21,4,12,9, 54,234,12,54,89,34,12,34,64,3,12,76,34,9,54,23,65,54,12,59,22};
//	int array_size = sizeof(array) / sizeof(int);
//	int array_byte_size = array_size * sizeof(int);
//	printf("The arrays size is : %d\nThe array byte size is : %d\nThe arrays is : \n", array_size, array_byte_size);
//	for (int i = 0; i < array_size; i++)
//		printf("%d\t", array[i]);
//	printf("\n");
//
//	dim3 block(4);
//	dim3 grid(4,2);
//
//	int* d_array;
//	cudaMalloc((void**)&d_array, array_byte_size);
//	cudaMemcpy(d_array, array, array_byte_size, cudaMemcpyHostToDevice);
//
//	thread_excersise_2d << <grid, block>> > (d_array);
//
//	return 0;
//		
//}