#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"



__global__ void no_divergence()
{
	int gid_of_thread = threadIdx.x + (blockIdx.x * blockDim.x);
	
	int warp_id_of_thread = gid_of_thread / 32;

	float a, b;
	a = b = 0;

	//here we will be diverging a warp and not a thread in a warp
	if (warp_id_of_thread % 2 == 0)
	{
		a = 100;
		b = 50.0;
	}
	else
	{
		a = 200;
		b = 75;
	}
}

__global__ void with_divergence()
{
	int gid_of_thread = threadIdx.x + (blockDim.x * blockIdx.x);

	float a, b;
	a = b = 0;


	//here we will be diverging the threads inside the warps
	if (gid_of_thread % 2 == 0)
	{
		a = 100;
		b = 50;
	}
	else
	{
		a = 200;
		b = 75;
	}
}

int main()
{
	printf("WARP DIVERGENCE EXAMPLE\n\n");


	dim3 grid();
	dim3 block(128);
}