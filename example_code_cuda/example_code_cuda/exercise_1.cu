//Print value of threadIdx, blockIdx, and gridDim variables for 3D grid which had 4 threads in all x, y, z dimensions and thread block size will be 2 threads in each dimension as well

#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void exercise_program1()
{
	printf("threadIdx.x : %d\tthreadIdx.y : %d\tthreadIdx.z : %d\t -- blockIdx.x : %d\tblockIdx.y : %d\tblockIdx.z : %d\t--- gridDim.x : %d\tgridDim.y : %d\tgridDim.z : %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

//int main()
//{
//	dim3 block(2,2,2);
//	dim3 grid(2, 2, 2);
//	exercise_program1 << <grid, block>> > ();
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}