#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>

//kernel decleration
__global__ void threadIdx_example()
{
	printf("threadIdx.x=\t%d, threadIdx.y=\t%d, threadIdx.z=\t%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	int nx=16, ny = 16;
	dim3 block(8, 8);
	dim3 grid(nx/block.x, ny/block.y);
	threadIdx_example << <grid, block>> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}