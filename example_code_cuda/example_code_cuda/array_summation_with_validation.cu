#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<random>


//kernel function
__global__ void array_summation_validation(int *a, int *b, int *c, int size)
{
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

void array_summation_validation_cpu(int* a_cpu, int* b_cpu, int* c_cpu, int size)
{
	for (int i = 0; i < size; i++)
	{
		c_cpu[i] = a_cpu[i] + b_cpu[i];
	}
}

void validation_function(int* gpu_result, int* cpu_result, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (gpu_result[i] != cpu_result[i])
		{
			printf("Arrays are not Equal!\n");
			return;
		}
	}
	printf("\n\nthe arrays are same and VALIDATED!\n");
}

int main()
{
	int array_size = 1000;
	int array_byte_size = array_size * sizeof(int);
	//host pointer variables to store the values of the arrays
	int *a1, *a2, *results;

	//assigning memory space for the variables and storing the head address of the array of memory
	a1 = (int*)malloc(array_byte_size);
	a2 = (int*)malloc(array_byte_size);
	results = (int*)malloc(array_byte_size);

	//assigning memory space for the variables for CPU generated results
	int* result_cpu;//, *a2_cpu, * a1_cpu;

	//allocating memory location for the cpu generated arrays
	result_cpu = (int*)malloc(array_byte_size);
	//a2_cpu = (int*)malloc(array_byte_size);
	//results_cpu = (int*)malloc(array_byte_size);

	//rnadomly initialize the array with random function
	for (int i = 0; i < array_size; i++)
	{
		a1[i] = rand() % 100+1;
		a2[i] = rand() % 100 + 1;
	}

	//printing all the variables of a1 and a2
	for (int i = 0; i < array_size; i++)
	{
		printf("a1[%d] = %d\ta2[%d] = %d\n", i, a1[i], i, a2[i]);
	}
	printf("\n");

	//declaring pointer variables in host to store the address of the device variables
	int *device_a1_address, *device_a2_address, *device_result_address;

	//allocating memory in device for a1, a2, and results
	hipMalloc((int**)&device_a1_address, array_byte_size);
	hipMalloc((int**)&device_a2_address, array_byte_size);
	hipMalloc((int**)&device_result_address, array_byte_size);

	//copying the  values for host variables to the device variables
	hipMemcpy(device_a1_address, a1, array_byte_size, hipMemcpyHostToDevice);
	hipMemcpy(device_a2_address, a2, array_byte_size, hipMemcpyHostToDevice);

	//grid and block dimensions
	int no_of_threads_per_block = 128;
	dim3 grid((array_size/no_of_threads_per_block + 1)); //+1 is to gaurantee that we will hae more grid than needed to avoid unwanted errors
	dim3 block(no_of_threads_per_block);

	//calling the kernel function
	array_summation_validation <<<grid, block>>> (device_a1_address, device_a2_address, device_result_address, array_size);
	hipDeviceSynchronize();

	//performing CPU array calculation
	array_summation_validation_cpu(a1, a2, result_cpu, array_size);

	//after kernel execution, we have to transfer the results from device to the host
	hipMemcpy(results, device_result_address, array_byte_size, hipMemcpyDeviceToHost);

	//function to validate the CPU and GPU calculation
	validation_function(results, result_cpu, array_size);

	//freeing memory in Host and device (Resetting device)
	hipFree(device_a1_address); hipFree(device_a2_address); hipFree(device_result_address);
	free(a1);					 free(a2);					  free(results);
	hipDeviceReset();
	return 0;
}
