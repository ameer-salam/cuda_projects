#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void uniqueIdx_example(int* input)
{
	int tid = threadIdx.x;
	int offset = blockDim.x * blockIdx.x;
	int gid = tid + offset;
	printf("threadIdx.x : %d,\t blockIdx.x : %d,\tgid : %d,\tvalue : %d\n",
			threadIdx.x, blockIdx.x, gid, input[gid]);
}

//int main()
//{
//	int array_size = 10;
//	int array_byte_size = sizeof(int) * array_size;
//	int array[] = { 6,2,8,9,2,31,0,95, 1, 4 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("%d  ", array[i]);
//	}
//	printf("\n\n");
//
//	int* a_data;
//	cudaMalloc((void**)&a_data, array_byte_size);
//	cudaMemcpy(a_data, array, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(5);
//	dim3 grid(2);
//
//
//	uniqueIdx_example << <grid, block>> > (a_data);
//	return 0;
//}

//output
/*
6  2  8  9  2  31  0  95  1  4

threadIdx.x : 0,         blockIdx.x : 1,        gid : 5,        value : 31
threadIdx.x : 1,         blockIdx.x : 1,        gid : 6,        value : 0
threadIdx.x : 2,         blockIdx.x : 1,        gid : 7,        value : 95
threadIdx.x : 3,         blockIdx.x : 1,        gid : 8,        value : 1
threadIdx.x : 4,         blockIdx.x : 1,        gid : 9,        value : 4
threadIdx.x : 0,         blockIdx.x : 0,        gid : 0,        value : 6
threadIdx.x : 1,         blockIdx.x : 0,        gid : 1,        value : 2
threadIdx.x : 2,         blockIdx.x : 0,        gid : 2,        value : 8
threadIdx.x : 3,         blockIdx.x : 0,        gid : 3,        value : 9
threadIdx.x : 4,         blockIdx.x : 0,        gid : 4,        value : 2
*/