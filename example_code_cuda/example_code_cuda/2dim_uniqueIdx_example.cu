#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

//__global__ void 2dim_uniqueIdx_example()
//{
//	int tidx = threadIdx.x;
//	int tidy = threadIdx.y;
//	int offsetx = blockIdx.x * blockDim.x;
//	int offsety = blockIdx.y * blockDim.y;
//	int gidx = tidx + offsetx;
//	int gidy = tidy + offsety;
//	printf("threadIdx : %d,\t")
//}

__global__ void unique_grid_calculation_2d(int* input)
{
	int tid = threadIdx.x;
	//int offset = blockDim.x * blockIdx.x;
	int gid = gridDim.x * (blockIdx.y * blockDim.x) //first statement of exuation to calculate row offset
		+ (blockIdx.x * blockDim.x) //block offset
		+ tid; //threadId in block
	printf("block_idx : %d,\tblockIdx.y : %d,\tthreadIdx.x : %d,\t, gid : %d,\tdata : %d\n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int data[] = { 6,39,1,3,53,0,32,25,76,4,23,12,05,65,21,1 };
//
//	int *d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, data, array_byte_size, cudaMemcpyHostToDevice); 
//
//	dim3 block(4);
//	dim3 grid(2,2);
//
//	unique_grid_calculation_2d << <grid, block>> > (d_data);
//
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}