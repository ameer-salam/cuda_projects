//question
/*
imagine you have randomly initialized 64 elements array and you are going to 
pass this array to your device as well. Launch a 3D grid as shown below.

* grid - x=4, y=4, z=4
* block x=2, y=2, z=2
*/

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void example2_function(int* input)
{
	int local_tid = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.z * blockDim.x * blockDim.y);
	int block_id = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
	int grid = local_tid + (block_id * blockDim.x * blockDim.y * blockDim.z);
	printf("threadIdx.x : %d,\tthreadIdx.y : %d,\tthreadIdx.z : %d,\tblockId : %d,\tvalue : %d\n", threadIdx.x, threadIdx.y, threadIdx.z, block_id, input[grid]);
}

int main()
{
	int array_size = 64;
	int array_byte_size = array_size * sizeof(int);
	int array[] = {
	0, 1, 2, 3, 4, 5, 6, 7,
	8, 9, 10, 11, 12, 13, 14, 15,
	16, 17, 18, 19, 20, 21, 22, 23,
	24, 25, 26, 27, 28, 29, 30, 31,
	32, 33, 34, 35, 36, 37, 38, 39,
	40, 41, 42, 43, 44, 45, 46, 47,
	48, 49, 50, 51, 52, 53, 54, 55,
	56, 57, 58, 59, 60, 61, 62, 63
	};

	dim3 block(2, 2, 2);
	dim3 grid(2, 2, 2);
	int* d_data;

	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, array, array_byte_size, hipMemcpyHostToDevice);

	example2_function <<<grid, block >>> (d_data);

	hipDeviceSynchronize();
	hipDeviceReset();

	return 0;
}