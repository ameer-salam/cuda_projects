#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


__global__ void warp_div_ex()
{
	int gid = threadIdx.x;
	int warpId = gid / 32;
	int even, odd;
	if (warpId == 0)
	{
		even += 2;
	}
	else
		odd += 1;
		//printf("threadIdx.x : %d\tpresent in the warp : %d\n", gid, warpId);
}

//int main()
//{
//	dim3 grid(1);
//	dim3 block(64);
//
//	warp_div_ex <<<grid, block>> > ();
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}