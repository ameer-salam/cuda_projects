
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//
//__global__ void dim_var_ex()
//{
//	printf("Hello CUDA!\n");
//}
//
//int main()
//{
//	dim3 block(4, 1, 1);
//	dim3 grid(8, 1, 1);
//	dim_var_ex <<< grid, block >>>();
//
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//
//
//	return 0;
//}