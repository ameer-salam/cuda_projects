
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//
//__global__ void dim_var_ex()
//{
//	printf("Hello CUDA!\n");
//}
//
//int main()
//{
//	dim3 block(8, 2, 1);
//	dim3 grid(2, 2, 1);
//	dim_var_ex << < grid, block >> > ();
//
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//
//
//	return 0;
//}
//
