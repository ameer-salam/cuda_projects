#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void rando_number_(int* input, int size)
{
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);
	if(gid < size)
	printf("threadIdx.x : %d.\tblockIdx.x : %d,\tgid : %d\tValue : %d\n", threadIdx.x, blockIdx.x, gid, input[gid]);
}

int main()
{
	int size = 150;
	int byte_size = size * sizeof(int);

	int* h_data;
	h_data = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
		h_data[i] = (int)(rand() & 0xff);
	int* d_input;
	hipMalloc((void**)&d_input, byte_size);
	hipMemcpy(d_input, h_data, byte_size, hipMemcpyHostToDevice);

	dim3 block(32);
	dim3 grid(5);

	rando_number_ << <grid, block >> > (d_input, size);

	hipDeviceSynchronize();
	hipDeviceReset();

	return 0;
}

