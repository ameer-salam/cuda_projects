#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void blockIdx_example()
{
	printf("blockIdx.x : %d\tblockIdx.y : %d\tblockIdx.z : %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
}

//int main()
//{
//	int nx = 16, ny = 16;
//	dim3 block(8, 8);
//	dim3 grid(nx / block.x, ny / block.y);
//	blockIdx_example << <grid, block>> > ();
//	cudaDeviceSynchronize();
//	cudaDeviceReset();
//	return 0;
//}