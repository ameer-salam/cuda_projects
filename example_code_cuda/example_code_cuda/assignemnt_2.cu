#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <random>
#include <chrono>

// CPU addition function
void cpu_addition(int* a, int* b, int* c, int* results, int size)
{
    for (int i = 0; i < size; i++)
    {
        results[i] = a[i] + b[i] + c[i];
    }
    printf("CPU calculations done!\n");
}

// GPU addition function
__global__ void gpu_addition(int* a, int* b, int* c, int* d, int size)
{
    int gid = threadIdx.x + (blockIdx.x * blockDim.x);
    if (gid < size)
    {
        d[gid] = a[gid] + b[gid] + c[gid];
    }
}

void compare(int* a, int* b, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
        {
            printf("They do not match!");
            return;
        }
    }
    printf("\nThe arrays Match!\n\n");
    int randno = rand() % size + 1;
    printf("Example : cpu_result[%d] = %d\tgpu_result[%d] = %d\n", randno, a[randno], randno, b[randno]);
    randno = rand() % size + 1;
    printf("Example : cpu_result[%d] = %d\tgpu_result[%d] = %d\n", randno, a[randno], randno, b[randno]);
}

//int main()
//{
//    int array_size = 4194304;
//    int array_byte_size = 4194304 * sizeof(int);
//    int* a1, * a2, * a3, * c_result, * g_results;
//    cudaError error;
//
//    // Allocating memory for the arrays
//    a1 = (int*)malloc(array_byte_size);
//    a2 = (int*)malloc(array_byte_size);
//    a3 = (int*)malloc(array_byte_size);
//    c_result = (int*)malloc(array_byte_size);
//    g_results = (int*)malloc(array_byte_size);
//
//    // Randomly allocating elements to the arrays
//    for (int i = 0; i < array_size; i++)
//    {
//        a1[i] = rand() % array_size + 1;
//        a2[i] = rand() % array_size + 1;
//        a3[i] = rand() % array_size + 1;
//    }
//
//    // CPU addition function
//    auto cpu_start = std::chrono::high_resolution_clock::now();
//    cpu_addition(a1, a2, a3, c_result, array_size);
//    auto cpu_stop = std::chrono::high_resolution_clock::now();
//    std::chrono::duration<double> cpu_duration = cpu_stop - cpu_start;
//
//    // Preparing for the kernel function
//    // Declaring pointer variables to point to the device memory
//    int* d_a1, * d_a2, * d_a3, * d_results;
//
//    // Allocating memory to store the variables on the device
//    error = cudaMalloc((void**)&d_a1, array_byte_size);
//    error = cudaMalloc((void**)&d_a2, array_byte_size);
//    error = cudaMalloc((void**)&d_a3, array_byte_size);
//    error = cudaMalloc((void**)&d_results, array_byte_size);
//
//    // Transferring data from host to the device
//    cudaMemcpy(d_a1, a1, array_byte_size, cudaMemcpyHostToDevice);
//    cudaMemcpy(d_a2, a2, array_byte_size, cudaMemcpyHostToDevice);
//    cudaMemcpy(d_a3, a3, array_byte_size, cudaMemcpyHostToDevice);
//
//    // Declaring the grid and block
//    int block_size = 512;
//    dim3 block(block_size);
//    dim3 grid(array_size / block_size);
//
//    // Using cudaEvent to measure GPU time
//    cudaEvent_t start, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//
//    // Starting GPU timing
//    cudaEventRecord(start);
//    gpu_addition << <grid, block >> > (d_a1, d_a2, d_a3, d_results, array_size);
//    cudaEventRecord(stop);
//
//    // Waiting for the event to complete
//    cudaEventSynchronize(stop);
//
//    // Calculating elapsed time
//    float milliseconds = 0;
//    cudaEventElapsedTime(&milliseconds, start, stop);
//
//    // Copying back the result from GPU to CPU
//    cudaMemcpy(g_results, d_results, array_byte_size, cudaMemcpyDeviceToHost);
//
//    // Comparison function
//    compare(c_result, g_results, array_size);
//
//    // Printing the time
//    printf("CPU time: %4.16f seconds\n", cpu_duration.count());
//    printf("GPU time: %4.16f seconds\n", milliseconds / 1000.0);
//
//    // Freeing memory
//    free(a1); free(a2); free(a3); free(c_result); free(g_results);
//    cudaFree(d_a1); cudaFree(d_a2); cudaFree(d_a3); cudaFree(d_results);
//    cudaEventDestroy(start);
//    cudaEventDestroy(stop);
//    cudaDeviceReset();
//
//    return 0;
//}
