#include "hip/hip_runtime.h"


#include <stdio.h>


//for two dimensional grid
__global__ void thread_excersise_2d_2d(int* input)
{
	int tid = blockDim.x * threadIdx.y + threadIdx.x;

	int threads_in_block = blockDim.x * blockDim.y;
	int block_offset = (blockIdx.x * threads_in_block);

	int threads_in_row = threads_in_block * gridDim.x;
	int row_offset = threads_in_row * blockIdx.y;

	int gid = tid + (block_offset + row_offset);
	printf("threadIdx.x : %d,\t, blockIdx.x : %d,\tblockIdx.y : %d,\tgid : %d,\tvalue : %d\n", tid, blockIdx.x, blockIdx.y, gid, input[gid]);
}

//int main()
//{
//	int array[] = { 77,43,5,3,2,8,34,21,4,12,9, 54,234,12,54,89,34,12,34,64,3,12,76,34,9,54,23,65,54,12,59,22 };
//	int array_size = sizeof(array) / sizeof(int);
//	int array_byte_size = array_size * sizeof(int);
//	printf("The arrays size is : %d\nThe array byte size is : %d\nThe arrays is : \n", array_size, array_byte_size);
//	for (int i = 0; i < array_size; i++)
//		printf("%d\t", array[i]);
//	printf("\n");
//
//	dim3 block(2,2);
//	dim3 grid(4, 2);
//
//	int* d_array;
//	cudaMalloc((void**)&d_array, array_byte_size);
//	cudaMemcpy(d_array, array, array_byte_size, cudaMemcpyHostToDevice);
//
//	thread_excersise_2d_2d << <grid, block >> > (d_array);
//
//	return 0;
//
//}