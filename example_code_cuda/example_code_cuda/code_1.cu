#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>

//declaring the kernel function
__global__ void hello_cuda()
{
	printf("Hello CUDA!");
}

int main()
{
	hello_cuda << <1, 1 >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();  //to reset the device and the memory too
	return 0;
}